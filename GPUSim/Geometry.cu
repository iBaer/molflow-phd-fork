#include "hip/hip_runtime.h"
// Created by pbahr

#include <optix_device.h>
#include "LaunchParams.h"

#define DET33(_11,_12,_13,_21,_22,_23,_31,_32,_33)  \
  ((_11)*( (_22)*(_33) - (_32)*(_23) ) +            \
   (_12)*( (_23)*(_31) - (_33)*(_21) ) +            \
   (_13)*( (_21)*(_32) - (_31)*(_22) ))

#define DOT(v1, v2)  \
  ((v1.x)*(v2.x) + (v1.y)*(v2.y) + (v1.z)*(v2.z))

#define CROSS(a, b)   \
  (a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x)

#define float3_as_args(u) \
    reinterpret_cast<cuuint32_t&>((u).x), \
    reinterpret_cast<cuuint32_t&>((u).y), \
    reinterpret_cast<cuuint32_t&>((u).z)


using namespace flowgpu;

namespace flowgpu {

    /*! launch parameters in constant memory, filled in by optix upon
        optixLaunch (this gets filled in from the buffer we pass to
        optixLaunch) */
    extern "C" __constant__ LaunchParams optixLaunchParams;

    //------------------------------------------------------------------------------
    // closest hit and anyhit programs for radiance-type rays.
    //
    // Note eventually we will have to create one pair of those for each
    // ray type and each geometry type we want to render; but this
    // simple example doesn't use any actual geometries yet, so we only
    // create a single, dummy, set of them (we do have to have at least
    // one group of them to set up the SBT)
    //------------------------------------------------------------------------------

    // Parallelogram intersection from the SDK optixWhitted example
    extern "C" __device__ void intersection__parallelogram__camera()
    {
        //const Parallelogram* floor = reinterpret_cast<Parallelogram*>( optixGetSbtDataPointer() );
        const PolygonMeshSBTData &sbtData = *(const PolygonMeshSBTData*)optixGetSbtDataPointer();

        const int   primID = optixGetPrimitiveIndex();
        const Polygon& poly  = sbtData.poly[primID];
        const vec3f &Aa     = sbtData.vertex[sbtData.index3[poly.vertOffset + 0]];
        const vec3f &Bb     = sbtData.vertex[sbtData.index3[poly.vertOffset + 1]];
        const vec3f &Cc     = sbtData.vertex[sbtData.index3[poly.vertOffset + 2]];

        vec3f v1 = Bb-Aa; // v1 = P0P1
        vec3f v2 = Cc-Aa; // v2 = P1P2
        vec3f n = cross(v1,v2);
        v1 *= 1.0f / dot( v1, v1 );
        v2 *= 1.0f / dot( v2, v2 );

        //printf("Normal (on device): %10.4f %10.4f %10.4f \n", n.x, n.y, n.z);
        //f->sh.N = CrossProduct(v1, v2);
        //vec3f n = cross(A,B);

        /*int ind = 2;
        while (ind < poly.nbVertices) {
            int i2 = sbtData.index[poly.vertOffset+ind++];

            v1 = Bb - Aa; // v1 = P0P1
            v2 = sbtData.vertex[i2] - Bb; // v2 = P1P2
            n = cross(v1, v2);              // Cross product
        }*/

        //n = poly.Nuv;
        //v1 = poly.U;
        //v2 = poly.V;

        vec3f ray_orig = optixGetWorldRayOrigin();
        vec3f ray_dir  = optixGetWorldRayDirection();

        const float ray_tmin = optixGetRayTmin(), ray_tmax = optixGetRayTmax();

        //make_float3( floor->plane );
        float dt = dot(ray_dir, n );
        float t = (dot(n,Aa) - dot(n, ray_orig))/dt;
        if( t > ray_tmin && t < ray_tmax )
        {
            vec3f p = ray_orig + ray_dir * t;
            vec3f vi = p - Aa;
            float a1 = dot(v1, vi);
            if(a1 >= 0 && a1 <= 1)
            {
                float a2 = dot(v2, vi);
                if(a2 >= 0 && a2 <= 1)
                {
                    //intersection__polygon(t,a1,a2,n);
                    optixReportIntersection(
                            t,
                            0,
                            float3_as_args(n),
                            float_as_int( a1 ), float_as_int( a2 )
                    );
                }
            }
        }
    }

    extern "C" __device__ void intersection__polygon(float d, float u, float v, vec3f n) {
        // Fast method to check if a point is inside a polygon or not.
        // Works with convex and concave polys, orientation independent

        const PolygonMeshSBTData &sbtData = *(const PolygonMeshSBTData*)optixGetSbtDataPointer();
        const int   primID = optixGetPrimitiveIndex();

        const Polygon& poly  = sbtData.poly[primID];

        const int nbSizeMinusOne = poly.nbVertices - 1;
        const vec2f* polyPoints = sbtData.vertex2;

        int n_updown = 0;
        int n_found = 0;

        vec2f p;
        p.u = u;
        p.v = v;

        for (size_t j = 0; j < nbSizeMinusOne; j++) {
            const vec2f& p1 = polyPoints[poly.vertOffset+j];
            const vec2f& p2 = polyPoints[poly.vertOffset+j+1];
/*
            if(primID==2 && optixGetLaunchIndex().x+optixGetLaunchIndex().y*optixLaunchParams.frame.size.x % 500 == 0)
                */
/*printf("[%d] -- %10.4f / %10.4f / %10.4f / %10.4f > %10.4f for Ray from %10.4f , %10.4f , %10.4f to %10.4f , %10.4f , %10.4f \n",
                       primID, det, u, v, d, ray_tmin, ray_orig.x,ray_orig.y,ray_orig.z,ray_dir.x,ray_dir.y,ray_dir.z);*//*

                printf("[%d] -- %10.4f , %10.4f -- %10.4f , %10.4f \n", sbtData.index[poly.vertOffset+j], p1.x, p1.y, p2.x, p2.y);
*/

            if (p.u<p1.u != p.u<p2.u) {
                float slope = (p2.v - p1.v) / (p2.u - p1.u);
                if ((slope * p.u - p.v) < (slope * p1.u - p1.v)) {
                    n_updown++;
                }
                else {
                    n_updown--;
                }
                n_found++;
            }
        }

/*
        if(primID==2 && optixGetLaunchIndex().x+optixGetLaunchIndex().y*optixLaunchParams.frame.size.x % 500 == 0)
            */
/*printf("[%d] -- %10.4f / %10.4f / %10.4f / %10.4f > %10.4f for Ray from %10.4f , %10.4f , %10.4f to %10.4f , %10.4f , %10.4f \n",
                   primID, det, u, v, d, ray_tmin, ray_orig.x,ray_orig.y,ray_orig.z,ray_dir.x,ray_dir.y,ray_dir.z);*//*

            printf("[%d]half -- found would be %d with %d and %d \n",nbSizeMinusOne, ((n_found / 2) & 1) ^ ((n_updown / 2) & 1), n_found, n_updown);
*/

        //Last point. Repeating code because it's the fastest and this function is heavily used
        const vec2f& p1 = polyPoints[poly.vertOffset+nbSizeMinusOne];
        const vec2f& p2 = polyPoints[poly.vertOffset+0];
        if (p.u<p1.u != p.u<p2.u) {
            float slope = (p2.v - p1.v) / (p2.u - p1.u);
            if ((slope * p.u - p.v) < (slope * p1.u - p1.v)) {
                n_updown++;
            }
            else {
                n_updown--;
            }
            n_found++;
        }

        /*if(primID==2 && optixGetLaunchIndex().x+optixGetLaunchIndex().y*optixLaunchParams.frame.size.x % 500 == 0)
            *//*printf("[%d] -- %10.4f / %10.4f / %10.4f / %10.4f > %10.4f for Ray from %10.4f , %10.4f , %10.4f to %10.4f , %10.4f , %10.4f \n",
                   primID, det, u, v, d, ray_tmin, ray_orig.x,ray_orig.y,ray_orig.z,ray_dir.x,ray_dir.y,ray_dir.z);*//*
            printf("[%d] -- found would be %d with %d and %d \n",nbSizeMinusOne, ((n_found / 2) & 1) ^ ((n_updown / 2) & 1), n_found, n_updown);*/
        if(((n_found / 2) & 1) ^ ((n_updown / 2) & 1)){
            optixReportIntersection(
                    d,
                    0,
                    float3_as_args(n),
                    float_as_int( u ), float_as_int( v )
            );
        }
    }

    // Parallelogram intersection based on the SDK optixWhitted example
    extern "C" __device__ void intersection__parallelogram()
    {
        const PolygonMeshSBTData &sbtData = *(const PolygonMeshSBTData*)optixGetSbtDataPointer();

        const int   primID = optixGetPrimitiveIndex();

        const vec3f ray_orig = optixGetWorldRayOrigin();
        vec3f ray_dir = optixGetWorldRayDirection();
        ray_dir = vec3f(-1.0,-1.0,-1.0) * ray_dir;

        const float ray_tmin = optixGetRayTmin(), ray_tmax = optixGetRayTmax();

        const Polygon& poly  = sbtData.poly[primID];
        const float det = dot(poly.Nuv, ray_dir);

        if(det > 0.0) {
            const float iDet = 1.0 / det;
            vec3f intZ = ray_orig - poly.O;
            const float u = iDet * DET33(intZ.x, poly.V.x, ray_dir.x,
                                   intZ.y, poly.V.y, ray_dir.y,
                                   intZ.z, poly.V.z, ray_dir.z);

            if (u >= 0.0 && u <= 1.0) {

                const float v = iDet * DET33(poly.U.x, intZ.x, ray_dir.x,
                                 poly.U.y, intZ.y, ray_dir.y,
                                 poly.U.z, intZ.z, ray_dir.z);

                if (v >= 0.0 && v <= 1.0) {

                    const float d = iDet * dot(poly.Nuv, intZ);

                    /*if(primID==2)
                        printf("[%d] -- %10.4f / %10.4f / %10.4f / %10.4f > %10.4f for Ray from %10.4f , %10.4f , %10.4f to %10.4f , %10.4f , %10.4f \n",
                               primID, det, u, v, d, ray_tmin, ray_orig.x,ray_orig.y,ray_orig.z,ray_dir.x,ray_dir.y,ray_dir.z);
*/
                    if (d>ray_tmin) {
                        //intersection__polygon(u,v);
                        intersection__polygon(d,u,v,poly.Nuv);
                        /*if(inPoly > 0){
                            optixReportIntersection(
                                    d,
                                    0,
                                    //float_as_int( poly.Nuv.x ), float_as_int( poly.Nuv.y ), float_as_int( n.z ),
                                    float3_as_args(poly.Nuv),
                                    float_as_int( u ), float_as_int( v )
                            );
                        }*/
                    }
                }
            }

        }
    }

    extern "C" __global__ void __intersection__polygon()
    {
        intersection__parallelogram();
    }

} // ::flowgpu
