#include "hip/hip_runtime.h"
//
// Created by pbahr on 04/11/2019.
//

#include <hip/hip_runtime.h>
#include "cudaRandom.cuh"
#include <hip/hip_math_constants.h>
#include <time.h>
#include <hiprand.h>
#include <stdio.h>
#include <stdlib.h>
#include "GPUDefines.h" // for NB_RAND

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states) {

    /* we have to initialize the state */
    const int id = threadIdx.x + blockIdx.x * 1;
    hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
                id, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
                0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                &states[id]);
}

/* this GPU kernel takes an array of states, and an array of ints, and puts a random int into each */
__global__ void randoms_bits(hiprandState_t* states, unsigned int* numbers) {
    /* hiprand works like rand - except that it takes a state as a parameter */
    numbers[blockIdx.x] = hiprand(&states[blockIdx.x]) % 100;
}

/* this GPU kernel takes an array of states, and an array of ints, and puts a random int into each */
__global__ void random_floats(hiprandState_t* states, float* numbers) {
    const int id = threadIdx.x + blockIdx.x * 1;
    /* Copy state to local memory for efficiency */
    hiprandState localState = states[id];
    /* hiprand works like rand - except that it takes a state as a parameter */
    for(int offset = 0; offset < NB_RAND; offset++)
        numbers[id + offset] = hiprand_uniform(&localState);

    /* Copy state back to global memory */
    states[id] = localState;
}

namespace crng {
/*
    cudaRandom::cudaRandom(unsigned int N) : kernelSize(N) {
        initializeRand();
    };

    cudaRandom::~cudaRandom() {
        */
/* free the memory we allocated for the states and numbers *//*

        hipFree(states);
        hipFree(randomNumbers);
    };
*/
    hiprandGenerator_t gen;
    int initializeRandHost(unsigned int kernelSize, float **randomNumbersPtr, unsigned int seed) {
        //hiprandGenerator_t gen;
        /*float *//**devData,*//* *hostData;

        *//* Allocate n floats on host *//*
        hostData = (float *)calloc(n, sizeof(float));*/

        /*size_t available, total;
        hipMemGetInfo(&available, &total);
        printf("Pre Available %d / %d\n",available,total);
        printf("Trying to allocate %d (Bytes)\n",NB_RAND*kernelSize*sizeof(float));*/

        /* Allocate n floats on device */
        CUDA_CALL(hipMalloc((void **)randomNumbersPtr, NB_RAND*kernelSize*sizeof(float)));
        //printf("Allocating size for %d random number\n",NB_RAND*kernelSize);
        /* Create pseudo-random number generator */
        //CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
        CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW));

        /* Set seed */
        CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, seed));

        return EXIT_SUCCESS;
    }

    int generateRandHost(unsigned int kernelSize, float *randomNumbers){
        /* Generate n floats on device */
        CURAND_CALL(hiprandGenerateUniform(gen, randomNumbers, NB_RAND*kernelSize));
        return EXIT_SUCCESS;
    }

    int destroyRandHost(float **randomNumbersPtr){
        /* Cleanup */
        CURAND_CALL(hiprandDestroyGenerator(gen));
        CUDA_CALL(hipFree(*randomNumbersPtr));
        return EXIT_SUCCESS;
    }

    int offsetBufferZeroInit(unsigned int kernelSize, void *randomOffsets){
        /* Generate n floats on device */
        CUDA_CALL(hipMemset((cuuint32_t*)randomOffsets, 0, kernelSize*sizeof(cuuint32_t)));
        return EXIT_SUCCESS;
    }

    int testRand(void** devData, size_t n){
        //size_t n = 100;
        hiprandGenerator_t gen;
        /*float *//**devData,*//* *hostData;

        *//* Allocate n floats on host *//*
        hostData = (float *)calloc(n, sizeof(float));*/

        /* Allocate n floats on device */
        CUDA_CALL(hipMalloc((void **)devData, n*sizeof(float)));

        /* Create pseudo-random number generator */
        CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));

        /* Set seed */
        CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen,
                                                       1234ULL));

        /* Generate n floats on device */
        CURAND_CALL(hiprandGenerateUniform(gen, (float*)*devData, n));

        /* Copy device memory to host *//*
        CUDA_CALL(hipMemcpy(hostData, devData, n * sizeof(float),
                             hipMemcpyDeviceToHost));

        *//* Show result *//*
        for(i = 0; i < n; i++) {
            if(i%100==0)
            printf("[%d] -- %1.4f ", i, hostData[i]);
        }
        printf("\n");*/

        /* Cleanup */
        CURAND_CALL(hiprandDestroyGenerator(gen));
        //CUDA_CALL(hipFree(devData));
/*
        free(hostData);
*/


        return EXIT_SUCCESS;
    }

    int printDevDataAtHost(void* devData, size_t n){
        float *hostData;

        /* Allocate n floats on host */
        hostData = (float *)calloc(n, sizeof(float));

        /* Copy device memory to host */
        CUDA_CALL(hipMemcpy(hostData, devData, n * sizeof(float), hipMemcpyDeviceToHost));

        /* Show result */
        cuuint32_t countSub = 0;
        cuuint32_t countSuper = 0;

        size_t i;
        for(i = 0; i < n; i++) {
            //if(i%100==0)
                //printf("[%d] -- %1.4f ", i, hostData[i]);
            if(i%100<10){
            if(hostData[i]<0.03)
            printf("[%zd] -- %1.4f ", i, hostData[i]);
            if(i%100==0)
                printf("\n");
            if(hostData[i]<0.03)
                countSub++;
            else
                countSuper++;}
        }
        printf("\n");
        printf("[%6.4f/%d -- %6.4f/%d] ", (float)(countSub)/(countSub+countSuper),countSub, (float)(countSuper)/(countSub+countSuper),countSuper);
        printf("\n");
        free(hostData);
        return EXIT_SUCCESS;
    }

    int  initializeRand(unsigned int kernelSize, void* states, void* randomNumbers) {
        /* CUDA's random number library uses hiprandState_t to keep track of the seed value
         we will store a random state for every thread  */

        /* allocate space on the GPU for the random states */
        CUDA_CALL(hipMalloc((void **) &states, kernelSize * sizeof(hiprandState_t)));

        unsigned int seed = time(0);
        /* we have to initialize the state */
        init << < kernelSize, 1 >> > (seed, (hiprandState_t*) states);

        CUDA_CALL(hipMalloc((void **) &randomNumbers, NB_RAND * kernelSize * sizeof(float))); // 10 rand per thread
        /* Set results to 0 */
        CUDA_CALL(hipMemset((float *) randomNumbers, 0.0f, NB_RAND * kernelSize *sizeof(float)));

        return EXIT_SUCCESS;
    }

    int  initializeRand(unsigned int kernelSize, hiprandState_t *states, float *randomNumbers) {
        /* CUDA's random number library uses hiprandState_t to keep track of the seed value
         we will store a random state for every thread  */

        /* allocate space on the GPU for the random states */
        CUDA_CALL(hipMalloc((void **) &states, kernelSize * sizeof(hiprandState_t)));

        unsigned int seed = time(0);
        /* we have to initialize the state */
        init << < kernelSize, 1 >> > (seed, states);

        CUDA_CALL(hipMalloc((void **) &randomNumbers, NB_RAND * kernelSize * sizeof(float))); // 10 rand per thread
        /* Set results to 0 */
        CUDA_CALL(hipMemset(randomNumbers, 0.0f, NB_RAND * kernelSize *sizeof(float)));

        return EXIT_SUCCESS;
    }

    void generateRand(unsigned int kernelSize, hiprandState_t *states, float *randomNumbers) {
        /* invoke the kernel to get some random numbers */
        random_floats << < kernelSize, 1 >> > (states, randomNumbers);
    }

    void destroyRand(hiprandState_t *states, float *randomNumbers){
        hipFree(states);
        hipFree(randomNumbers);
    }
}
